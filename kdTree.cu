#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <windows.h>
#include <time.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <thrust\sort.h>
#include <thrust\scan.h>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\fill.h> 
#include <thrust\sequence.h>
#include <thrust\copy.h>



template<typename Argument1, typename Argument2, typename Result> struct binary_function  : public std::binary_function<Argument1, Argument2, Result>{};



//timer stuff
double PCFreq = 0.0;
__int64 CounterStart = 0;

//yeah it's really 112 or 480 (for me), but for testing 512 is easier
#define numOfCudaCores = 512;

__global__ void merge_x(float3* dPoints, int size)
{	
	int sizeOfSubArray=size;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int i = x * sizeOfSubArray;
	int n = i + sizeOfSubArray;
	int m = 1;
	//bottom-up merge sort
	while(m<=n)
	{
		i = x * sizeOfSubArray;
		while(i<(n-m))
		{
			int endPos = (i+2*m-1) > (n-1) ? (n-1) : (i+2*m-1);
			//the merging part aka insertion
			int lenSubArr = (i + (endPos - i));
			int w = i;
			while(w<lenSubArr)
			{
				w=i;
				for(int r = (i+1); r <= lenSubArr; r++)
				{
					if(dPoints[w].x<dPoints[r].x)
					{
						w=r;
					}
				}
				float temp1 = dPoints[w].x;
				float temp2 = dPoints[w].y;
				float temp3 = dPoints[w].z;
				dPoints[w].x=dPoints[lenSubArr].x;
				dPoints[w].y=dPoints[lenSubArr].y;
				dPoints[w].z=dPoints[lenSubArr].z;
				dPoints[lenSubArr].x=temp1;
				dPoints[lenSubArr].y=temp2;
				dPoints[lenSubArr].z=temp3;
				lenSubArr--;
			}
			i = i + 2 * m;
		}
		m = m * 2;
	}	
}

__global__ void merge_y(float3* dPoints, int size)
{	
	int sizeOfSubArray=size;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int i = x * sizeOfSubArray;
	int n = i + sizeOfSubArray;
	int m = 1;
	//bottom-up merge sort
	while(m<=n)
	{
		i = x * sizeOfSubArray;
		while(i<(n-m))
		{
			int endPos = (i+2*m-1) > (n-1) ? (n-1) : (i+2*m-1);
			//the merging part aka insertion
			int lenSubArr = (i + (endPos - i));
			int w = i;
			while(w<lenSubArr)
			{
				w=i;
				for(int r = (i+1); r <= lenSubArr; r++)
				{
					if(dPoints[w].y<dPoints[r].y)
					{
						w=r;
					}
				}
				float temp1 = dPoints[w].x;
				float temp2 = dPoints[w].y;
				float temp3 = dPoints[w].z;
				dPoints[w].x=dPoints[lenSubArr].x;
				dPoints[w].y=dPoints[lenSubArr].y;
				dPoints[w].z=dPoints[lenSubArr].z;
				dPoints[lenSubArr].x=temp1;
				dPoints[lenSubArr].y=temp2;
				dPoints[lenSubArr].z=temp3;
				lenSubArr--;
			}
			i = i + 2 * m;
		}
		m = m * 2;
	}	
}

__global__ void merge_z(float3* dPoints, int size)
{	
	int sizeOfSubArray=size;
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int i = x * sizeOfSubArray;
	int n = i + sizeOfSubArray;
	int m = 1;
	//bottom-up merge sort
	while(m<=n)
	{
		i = x * sizeOfSubArray;
		while(i<(n-m))
		{
			int endPos = (i+2*m-1) > (n-1) ? (n-1) : (i+2*m-1);
			//the merging part aka insertion
			int lenSubArr = (i + (endPos - i));
			int w = i;
			while(w<lenSubArr)
			{
				w=i;
				for(int r = (i+1); r <= lenSubArr; r++)
				{
					if(dPoints[w].z<dPoints[r].z)
					{
						w=r;
					}
				}
				float temp1 = dPoints[w].x;
				float temp2 = dPoints[w].y;
				float temp3 = dPoints[w].z;
				dPoints[w].x=dPoints[lenSubArr].x;
				dPoints[w].y=dPoints[lenSubArr].y;
				dPoints[w].z=dPoints[lenSubArr].z;
				dPoints[lenSubArr].x=temp1;
				dPoints[lenSubArr].y=temp2;
				dPoints[lenSubArr].z=temp3;
				lenSubArr--;
			}
			i = i + 2 * m;
		}
		m = m * 2;
	}	
}

struct float3Array
{
	float* x;
	float* y;
	float* z;
	int* index;
};

struct node
{
	 float3 point;
	 int index;
	 node *parent;
	 node *leftChild;
	 node *rightChild;
};

struct compare_float3_x
{
	__host__ __device__
	bool operator()(float3 a, float3 b)
	{
		return a.x < b.x;
	}
};
struct compare_float3_y
{
	__host__ __device__
	bool operator()(float3 a, float3 b)
	{
		return a.y < b.y;
	}
};
struct compare_float3_z
{
	__host__ __device__
	bool operator()(float3 a, float3 b)
	{
		return a.z < b.z;
	}
};

//specialMaximum retuns the maximum value between two integers
//unless if they are equal, then it will return the integer++
//this will be used to update our SubArray
template<typename T>
struct specialMaximum : public thrust::binary_function<T,T,T>
{
	__host__ __device__ 
	const T operator()(const T &lhs, const T &rhs) const
	{
	  if(lhs<rhs)
	  {
		 return (((int)rhs)+1);
	  }
	  if(lhs==rhs)
	  {
		  if(lhs==0)
		  {
			  return 0;
		  }
		  return (((int)rhs)+1);
	  }
	  return lhs < rhs ? rhs : lhs;
  }
};


float3 make_random_float3(void);
void StartCounter();
double GetCounter();
int constructKD(thrust::device_vector<float3>& dPoints, int whichDim, int begin, int end,	compare_float3_x& comp_x, compare_float3_y& comp_y ,compare_float3_z& comp_z, int numLevels);

int main(int argc, char* argv[])
{	
	int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
	hipSetDevice(0);
	/*if(deviceCount>1)
	{
		//1=9800gt, 0=gtx480
		//we want to develop/debug on the non-primary device if possible (at least on Windows)
		//set value to 0 for release builds or comment this line out
		hipSetDevice(1);
	}*/
		
	int numOfTriangles = 0;
	int numOfVertexes = 0;
	std::string line;
	char* file = "teapot.obj";
	std::ifstream myfile (file);
	size_t found;
	size_t found2;
	size_t found3;
	size_t found4;
	
	if (myfile.is_open())
	{
		while ( myfile.good() )
		{
	      std::getline (myfile,line);
		  found=line.find("f");
		  found4=line.find("#");
		  found2=line.find("v");
		  found3=line.find("n");
		  if (found!=std::string::npos && (!(found4!=std::string::npos)))
		  {
			  numOfTriangles++;
		  }
		  if ((found2!=std::string::npos) && (!(found3!=std::string::npos)))
		  {
			  numOfVertexes++;
		  }

	    }
		myfile.close();
	}
    else printf("Unable to open file");

	//how many Points we will have
	int numOfPoints=numOfTriangles;
	numOfTriangles=numOfTriangles*3;
	numOfVertexes=numOfVertexes*3;
	
	//creating the arrays on host
	thrust::host_vector<float> vertexArr(numOfVertexes);
	thrust::host_vector<float> triangleArr(numOfTriangles);
	thrust::host_vector<int> hSubArray(numOfPoints);
	thrust::host_vector<float3> hPoints(numOfPoints);

	//generating random numbers
	//thrust::generate(hPoints.begin(), hPoints.end(), make_random_float3);
	//fill index array
	thrust::fill(hSubArray.begin(), hSubArray.end(), 0);

	std::ifstream myfile2 (file);
	int current=0;
	if (myfile2.is_open())
	{
		while ( myfile2.good() )
		{
	      std::getline (myfile2,line);
		  found2=line.find("v");
		  found3=line.find("n");
		  if ((found2!=std::string::npos) && (!(found3!=std::string::npos)))
		  {
			  found2=line.find_first_not_of(" ",1);
			  int temp = (int(found2));
			  found2=line.find_first_of(" ",temp);
			  int tempEnd = (int(found2));
			  found2=line.find_first_not_of(" ",tempEnd);
			  int temp2 = (int(found2));
			  found2=line.find_first_of(" ",temp2);
			  int temp2End = (int(found2));
			  found2=line.find_first_not_of(" ",temp2End);
			  int temp3 = (int(found2));

			  std::string test = line.substr(temp,(tempEnd-temp));	
			  vertexArr[current]= atof((char*)test.c_str());
			  current++;

			  std::string test2=line.substr(temp2,(temp2End-temp2));
			  vertexArr[current]=atof((char*)test2.c_str());
			  current++;

			  std::string test3=line.substr(temp3);
			  vertexArr[current]=atof((char*)test3.c_str());
			  current++;

		  }

	    }
		myfile2.close();
	}
    else printf("Unable to open file");

	std::ifstream myfile3 (file);
	current=0;
	if (myfile3.is_open())
	{
		while ( myfile3.good() )
		{
	      std::getline (myfile3,line);

		  found=line.find("f");
		  found4=line.find("#");
		  if (found!=std::string::npos && (!(found4!=std::string::npos)))
		  {
			  found=line.find_first_not_of(" ",1);
			  int temp = (int(found));
			  found=line.find_first_of(" ",temp);
			  int tempEnd = (int(found));
			  found=line.find_first_not_of(" ",tempEnd);
			  int temp2 = (int(found));
			  found=line.find_first_of(" ",temp2);
			  int temp2End = (int(found));
			  found2=line.find_first_not_of(" ",temp2End);
			  int temp3 = (int(found));

			  std::string test = line.substr(temp,(tempEnd-temp));	
			  int firstTri = (((atoi((char*)test.c_str()))-1)*3);
			  std::string test2=line.substr(temp2,(temp2End-temp2));
			  int secondTri = (((atoi((char*)test2.c_str()))-1)*3);
			  std::string test3=line.substr(temp3);
			  int thirdTri = (((atoi((char*)test3.c_str()))-1)*3);
			  
			  float midX = (vertexArr[firstTri]+vertexArr[secondTri]+vertexArr[thirdTri])/3;
			  float midY = (vertexArr[firstTri+1]+vertexArr[secondTri+1]+vertexArr[thirdTri+1])/3;
			  float midZ = (vertexArr[firstTri+2]+vertexArr[secondTri+2]+vertexArr[thirdTri+2])/3;

			  triangleArr[current]= midX;
			  current++;
			  triangleArr[current]= midY;
			  current++;
			  triangleArr[current]= midZ;
			  current++;
		  }
		}
		myfile3.close();
	}
	else printf("Unable to open file");


	for(int w=0;w<numOfTriangles;w+=3)
	{
		int t = w/3;
		hPoints[t].x=triangleArr[w];
		hPoints[t].y=triangleArr[w+1];
		hPoints[t].z=triangleArr[w+2];
	}
	 
	//transfering values to device
	thrust::device_vector<float3> dPoints=hPoints;
	thrust::device_vector<int> dSubArray=hSubArray;
	double elapsed_time=0;
	compare_float3_x comp_x;
	compare_float3_y comp_y;
	compare_float3_z comp_z;

	int numLevels = 3;

	//Normally we would have the next three lines of code
	//int totalLevels = ((int) log2(numOfPoints+0.0f));
	//int numLevels = ((int) log2(512.0f));//this means keep going until we hit 512 subArrays
	//int * ptr = thrust::raw_pointer_cast(&dPoints[0]);

	hipDeviceSynchronize();
	StartCounter();


	int whichDim = constructKD(dPoints, 0, 0, numOfPoints, comp_x, comp_y, comp_z, numLevels);
	/*
	int currentLevel=numLevels;
	int numPartitions = 0;
	int nBlocks =0;
	while(currentLevel<=maxLevel)
	{
		numPartitions = ((int) pow(2.0f,currentLevel+0.0f));
		nBlocks = numPartitions/numOfCudaCores + (numPartitions%numOfCudaCores == 0?0:1);
		switch(whichDim)
		{
		case 0:
			merge_x <<< nBlocks, numPartitions,0 >>> (ptr,numPartitions);
			whichDim=1;
			break;
		case 1:
			merge_y <<< nBlocks, numPartitions,0 >>> (ptr,numPartitions);
			whichDim=2;
			break;
		case 2:
			merge_z <<< nBlocks, numPartitions,0 >>> (ptr,numPartitions);
			whichDim=0;
			break;
		default:
			printf("You shouldn't be here; i.e. wrong case number");
			break;
		}
		currentLevel++;
	}
	*/

	hipDeviceSynchronize();
	printf("Time elapsed: %G seconds\n", GetCounter());
	thrust::copy(dPoints.begin(), dPoints.end(), hPoints.begin());
	std::ofstream myOut ("kdtree.will");
	int blockNum = (pow(2.0f,numLevels));
	int blockSize = numOfPoints/blockNum;
	myOut << "$ " << blockNum << "\n";
	myOut << "& " << blockSize << "\n";
	for(int i=0; i<numOfPoints;i++)
	{
		if (myOut.is_open())
		{
			if( (!(i==0)) && i%blockSize==0)
			{
				myOut << "# " << hPoints[i].x << " " << hPoints[i].y << " " << hPoints[i].z << "\n";
			}
			else
			{
				myOut << "! " << hPoints[i].x << " " << hPoints[i].y << " " << hPoints[i].z << "\n";
			}
		}
		 else std::cout << "Unable to open file\n";
	}
	myOut<< "\n";
	myOut.close();
	//RAWR END
}

//creates a float3 with three random numbers
float3 make_random_float3(void)
{
	return make_float3( rand()+(rand()/(RAND_MAX + 1.0f)), rand()+(rand()/(RAND_MAX + 1.0f)), rand()+(rand()/(RAND_MAX + 1.0f)));
}

//whichDim simply means which dimension we are sorting by, 0 = x, 1 = y, 2 = z
int constructKD(thrust::device_vector<float3>& dPoints, int whichDim, int begin, int end,	compare_float3_x& comp_x, compare_float3_y& comp_y ,compare_float3_z& comp_z, int numLevels)
{
	switch(whichDim)
	{
	case 0:
		thrust::sort(dPoints.begin()+begin, dPoints.begin()+end, comp_x);	
		break;
	case 1:
		thrust::sort(dPoints.begin()+begin, dPoints.begin()+end, comp_y);
		break;
	case 2:
		thrust::sort(dPoints.begin()+begin, dPoints.begin()+end, comp_z);
		break;
	default:
		printf("You shouldn't be here; i.e. wrong case number");
		break;
	}

	switch(whichDim)
	{
	case 0:
		whichDim=1;
		break;
	case 1:
		whichDim=2;
		break;
	case 2:
		whichDim=0;
		break;
	default:
		printf("You shouldn't be here; i.e. wrong case number");
		break;
	}
	
	numLevels--;
	int numOfPoints = end-begin;
	int lowerBound = ((int)numOfPoints/2)+begin;
	int upperBound = ((int)numOfPoints/2)+1+begin;
	int toReturn=0;
	if(numLevels>0)
	{		
		toReturn=constructKD(dPoints, whichDim, begin, lowerBound, comp_x, comp_y, comp_z, numLevels);
		toReturn=constructKD(dPoints, whichDim, upperBound, end, comp_x, comp_y, comp_z, numLevels);
	}
	toReturn=whichDim;
	return toReturn;
}

void StartCounter()
{
    LARGE_INTEGER li;
    if(!QueryPerformanceFrequency(&li))
	printf("QueryPerformanceFrequency failed!\n");

    //Below is for seconds
	PCFreq = double(li.QuadPart);
	//Below is for milliseconds
	//PCFreq = double(li.QuadPart)/1000.0;
	//Below is for microseconds
	//PCFreq = double(li.QuadPart)/1000000.0;

    QueryPerformanceCounter(&li);
    CounterStart = li.QuadPart;
}
double GetCounter()
{
    LARGE_INTEGER li;
    QueryPerformanceCounter(&li);
    return double(li.QuadPart-CounterStart)/PCFreq;
}